#include "hip/hip_runtime.h"
#include "k_means.h"

int N = 10000000;
int K = 4;
int N_THREADS = 50000;
int N_BLOCKS = 200;

using namespace std;

struct point
{
    float x;
    float y;
    int nCluster;
};

// Function where the array of points and the array of centroids are populated:
void init(struct point *allpoints, struct point *allcentroids, int K, int N)
{
    chrono::steady_clock::time_point begin = chrono::steady_clock::now();

    srand(10);
    for (int i = 0; i < N; i++)
    {
        allpoints[i].x = (float)rand() / RAND_MAX;
        allpoints[i].y = (float)rand() / RAND_MAX;
        allpoints[i].nCluster = -1;
    }
    for (int j = 0; j < K; j++)
    {
        allcentroids[j].x = allpoints[j].x;
        allcentroids[j].y = allpoints[j].y;
    }

    chrono::steady_clock::time_point end = chrono::steady_clock::now();
    cout << endl
         << "Sequential CPU execution (init): " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " microseconds" << endl
         << endl;
}

__device__ float determineDistance(struct point p, struct point c)
{
    float diff_x = (p.x) - (c.x);
    float diff_y = (p.y) - (c.y);
    float aux = diff_x * diff_x + diff_y * diff_y;
    return aux;
}

// Function where we determine if a point should change the cluster it is currently assign to
__global__ void update_cluster_points(struct point *allpoints, struct point *allcentroids, int K, int N) // int *points_changed,
{
    float diff_temp;     // Variável privada
    int newCluster = -1; // Variável privada
    float diff;          // Variável privada
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= 0 && id < N)
    {
        diff = 100;
        // calculates the distance between a certain point and all the centroids
        for (int j = 0; j < K; j++)
        {
            // calculates the distance
            diff_temp = determineDistance(allpoints[id], allcentroids[j]);

            if (diff_temp < diff)
            {
                diff = diff_temp;
                newCluster = j;
            }
        }
        if (allpoints[id].nCluster != newCluster)
        {
            allpoints[id].nCluster = newCluster; // não deve ter data races
            // atomicAdd(points_changed, 1);
        }
    }
}

__global__ void initialize_sums_and_size(int *size, float *SumX, float *SumY, int K)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= 0 && id < K)
    {
        SumX[id] = 0;
        SumY[id] = 0;
        size[id] = 0;
    }
}

__global__ void mean_sums(int *size, float *SumX, float *SumY, struct point *allcentroids, int K)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= 0 && id < K)
    {
        allcentroids[id].x = SumX[id] / size[id]; // não deve haver data races
        allcentroids[id].y = SumY[id] / size[id]; // não deve haver data races
    }
}

__global__ void print_data(int K, float *SumX, float *SumY, int *size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id == 0)
    {
        for (int i = 0; i < K; i++)
        {
            printf("%f - ", SumX[i]);
            printf("%f - ", SumY[i]);
            printf("%d\n", size[i]);
        }
    }
}

/*__syncthreads(); //__syncthreads sincroniza apenas as threads do mesmo bloco
                 // Não encontrei nenhuma solução para sincronizar todas as threads*/

__global__ void sum_all_points(struct point *allpoints, int *size, float *SumX, float *SumY, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= 0 && id < N)
    {
        int nCluster = allpoints[id].nCluster;
        atomicAdd(&SumX[nCluster], allpoints[id].x);
        // SumX[nCluster] += allpoints[id].x; // possíveis data races
        atomicAdd(&SumY[nCluster], allpoints[id].y);
        // SumY[nCluster] += allpoints[id].y; // possíveis data races
        atomicAdd(&size[nCluster], 1);
        // sizeA[nCluster]++; // possíveis data races
    }
}

int kmeans(int *lenClusters, struct point *array_points, struct point *array_centroids, int K, int N)
{

    // int points_changed = 0;
    int nIterations = 0;

    init(array_points, array_centroids, K, N);
    chrono::steady_clock::time_point begin = chrono::steady_clock::now();

    struct point *darray_points; // Array with all the points of this program, for the device memory
    hipMalloc((void **)&darray_points, sizeof(struct point) * N);
    hipMemcpy(darray_points, array_points, sizeof(struct point) * N, hipMemcpyHostToDevice);

    struct point *darray_centroids; // Array with all the centroids, for the device memory
    hipMalloc((void **)&darray_centroids, sizeof(struct point) * K);
    hipMemcpy(darray_centroids, array_centroids, sizeof(struct point) * K, hipMemcpyHostToDevice);

    int *dlenClusters;
    hipMalloc((void **)&dlenClusters, sizeof(int) * K);
    checkCUDAError("mem allocation");

    float *SumX;
    hipMalloc((void **)&SumX, sizeof(float) * K);

    float *SumY;
    hipMalloc((void **)&SumY, sizeof(float) * K);
    checkCUDAError("mem malloc");

    update_cluster_points<<<N_THREADS, N_BLOCKS>>>(darray_points, darray_centroids, K, N); //&points_changed,
    do
    {
        // points_changed = 0;
        initialize_sums_and_size<<<N_THREADS, N_BLOCKS>>>(dlenClusters, SumX, SumY, K);
        // print_data<<<N_THREADS, N_BLOCKS>>>(K, SumX, SumY, dlenClusters);
        sum_all_points<<<N_THREADS, N_BLOCKS>>>(darray_points, dlenClusters, SumX, SumY, N);
        mean_sums<<<N_THREADS, N_BLOCKS>>>(dlenClusters, SumX, SumY, darray_centroids, K);

        update_cluster_points<<<N_THREADS, N_BLOCKS>>>(darray_points, darray_centroids, K, N); //&points_changed,
        nIterations++;
    } while (nIterations != 21);

    checkCUDAError("error while");

    hipMemcpy(array_centroids, darray_centroids, sizeof(struct point) * K, hipMemcpyDeviceToHost);
    hipMemcpy(lenClusters, dlenClusters, sizeof(int) * K, hipMemcpyDeviceToHost);
    checkCUDAError("mem cpy 2");

    hipFree(darray_points);
    hipFree(darray_centroids);
    hipFree(dlenClusters);
    hipFree(SumX);
    hipFree(SumY);
    checkCUDAError("mem free");

    chrono::steady_clock::time_point end = chrono::steady_clock::now();
    cout << endl
         << "Sequential CPU execution (while): " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " microseconds" << endl
         << endl;

    return nIterations;
}

int main(int argc, char *argv[])
{
    if (argc == 4)
    {
        N = atoi(argv[1]);         // Número de Pontos
        K = atoi(argv[2]);         // Número de Clusters
        N_THREADS = atoi(argv[3]); // Número de Threads
    }
    else if (argc == 3)
    {
        N = atoi(argv[1]); // Número de Pontos
        K = atoi(argv[2]); // Número de Clusters
        N_THREADS = 1;     // Número de Threads
    }

    struct point *array_points = (struct point *)malloc(sizeof(struct point) * N);    // Array with all the points of this program
    struct point *array_centroids = (struct point *)malloc(sizeof(struct point) * K); // Array with all the centroids
    int *lenClusters = (int *)malloc(sizeof(int) * K);

    startKernelTime();
    int nIterations = kmeans(lenClusters, array_points, array_centroids, K, N);
    stopKernelTime();
    checkCUDAError("kernel invocation");

    printf("N = %d, K = %d\n", N, K);
    for (int i = 0; i < K; i++)
    {
        printf("Center: (%.3f,%.3f) : Size %d \n", array_centroids[i].x, array_centroids[i].y, lenClusters[i]);
    }
    printf("Iterations: %d\n", nIterations - 1);

    free(array_points);
    free(array_centroids);
    free(lenClusters);

    return 0;
}