#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <chrono>

#include "k_means.h"


#define N  10000;
#define K  4;
#define N_THREADS  100;
#define N_BLOCKS  100;

using namespace std;

struct point
{
    float x;
    float y;
    int nCluster;
};

// Function where the array of points and the array of centroids are populated:
void init(struct point allpoints[N], struct point allcentroids[K])
{
    chrono::steady_clock::time_point begin = chrono::steady_clock::now();

    srand(10);
    for (int i = 0; i < N; i++)
    {
        allpoints[i].x = (float)rand() / RAND_MAX;
        allpoints[i].y = (float)rand() / RAND_MAX;
        allpoints[i].nCluster = -1;
    }
    for (int j = 0; j < K; j++)
    {
        allcentroids[j].x = allpoints[j].x;
        allcentroids[j].y = allpoints[j].y;
    }

    chrono::steady_clock::time_point end = chrono::steady_clock::now();
	cout << endl << "Sequential CPU execution (init): " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " microseconds" << endl << endl;
}

float determineDistance(struct point p, struct point c)
{
    float diff_x = (p.x) - (c.x);
    float diff_y = (p.y) - (c.y);
    float aux = diff_x * diff_x + diff_y * diff_y;
    return aux;
}

// Function where we determine if a point should change the cluster it is currently assign to
__global__ 
int update_cluster_points(struct point *allpoints, struct point *allcentroids, int points_changed)
{

        float diff_temp;
        int newCluster = -1;
        float diff;
        int id = blockIdx.x * blockDim.x + threadIdx.x;

        if (id => 0 && id < N)
        {
            diff = 100;
            // calculates the distance between a certain point and all the centroids
            for (int j = 0; j < K; j++)
            {
                // calculates the distance
                diff_temp = determineDistance(allpoints[id], allcentroids[j]);

                if (diff_temp < diff)
                {
                    diff = diff_temp;
                    newCluster = j;
                }
            }
            if (allpoints[id].nCluster != newCluster)
            {
                allpoints[id].nCluster = newCluster;
                atomicAdd(&points_changed, 1);
            }
        }
    __syncthreads();

}

__global__ 
void determine_new_centroid(int *size, struct point *allpoints, struct point *allcentroids)
{
    __device__ float SumX[K];
    __device__ float SumY[K];
    __device__ int sizeA[K];

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id => 0 && id < K)
    {
        SumX[id] = 0;
        SumY[id] = 0;
        sizeA[id] = 0;
    }
__syncthreads();

    if (id => 0 && id < N)
    {
        int nCluster = allpoints[id].nCluster;
        atomicAdd(&SumX[nCluster], allpoints[id].x);
            // SumX[nCluster] += allpoints[id].x; // possíveis data races
        atomicAdd(&SumY[nCluster], allpoints[id].y);
            // SumY[nCluster] += allpoints[id].y; // possíveis data races
        atomicAdd(&sizeA[nCluster], 1);
            // sizeA[nCluster]++; // possíveis data races
    }

__syncthreads();

    // Calculates the mean of the coordinates of all the centroids
    if (id => 0 && id < K)
    {
        allcentroids[id].x = SumX[id] / sizeA[id]; // não deve haver data races
        allcentroids[id].y = SumY[id] / sizeA[id]; // não deve haver data races
        size[id] = sizeA[id];                      // não deve haver data races
    }

    return;
}

int kmeans(int lenClusters[K], struct point array_points[N], struct point array_centroids[K], int *dlenClusters, struct point *darray_points, struct point *darray_centroids)
{

    int points_changed;
    int nIterations = 0;

    init(array_points, array_centroids);
    chrono::steady_clock::time_point begin = chrono::steady_clock::now();

    hipMemcpy (darray_points, array_points, sizeof(struct point) * N, hipMemcpyHostToDevice);
    hipMemcpy (darray_centroids, array_centroids, sizeof(struct point) * K, hipMemcpyHostToDevice);

    points_changed = update_cluster_points <<< N_THREADS, N_BLOCKS >>> (darray_points, darray_centroids);
    do
    {
        determine_new_centroid <<< N_THREADS, N_BLOCKS >>> (dlenClusters, darray_points, darray_centroids);
        update_cluster_points <<< N_THREADS, N_BLOCKS >>> (darray_points, darray_centroids, points_changed);
        nIterations++;
    } while (nIterations != 21);

    hipMemcpy (array_centroids, darray_centroids, sizeof(struct point) * K, hipMemcpyDeviceToHost);
    hipMemcpy (lenClusters, dlenClusters, sizeof(int) * K, hipMemcpyDeviceToHost);

    chrono::steady_clock::time_point end = chrono::steady_clock::now();
    cout << endl << "Sequential CPU execution (while): " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << " microseconds" << endl << endl;


    return nIterations;
}

int main(int argc, char *argv[])
{
    if (argc == 4)
    {
        N = atoi(argv[1]);         // Número de Pontos
        K = atoi(argv[2]);         // Número de Clusters
        N_THREADS = atoi(argv[3]); // Número de Threads
    }
    else if (argc == 3)
    {
        N = atoi(argv[1]); // Número de Pontos
        K = atoi(argv[2]); // Número de Clusters
        N_THREADS = 1;     // Número de Threads
    }

    struct point *darray_points;    // Array with all the points of this program
    struct point *darray_centroids; // Array with all the centroids
    int *dlenClusters;

    struct point *array_points = malloc(sizeof(struct point) * N);    // Array with all the points of this program
    struct point *array_centroids = malloc(sizeof(struct point) * K); // Array with all the centroids
    int *lenClusters = malloc(sizeof(int) * K);


    hipMalloc ((void**) &darray_points, sizeof(struct point) * N);
    hipMalloc ((void**) &darray_centroids, sizeof(struct point) * K);
    hipMalloc ((void**) &dlenClusters, sizeof(int) * K);
    checkCUDAError("mem allocation");


    startKernelTime ();
    int nIterations = kmeans(lenClusters, array_points, array_centroids, dlenClusters, darray_points, darray_centroids);
    stopKernelTime ();
    checkCUDAError("kernel invocation");

    printf("N = %d, K = %d\n", N, K);
    for (int i = 0; i < K; i++)
    {
        printf("Center: (%.3f,%.3f) : Size %d \n", array_centroids[i].x, array_centroids[i].y, lenClusters[i]);
    }
    printf("Iterations: %d\n", nIterations - 1);

    free(array_points);
    free(array_centroids);
    free(lenClusters);

    hipFree(darray_points);
    hipFree(darray_centroids);
    hipFree(dlenClusters);
    checkCUDAError("mem free");
    return 0;
}